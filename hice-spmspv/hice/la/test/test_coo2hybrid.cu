#include "hip/hip_runtime.h"
// This is used to Test cusparse's csr2csc kernel.

#include <iostream>
#include <string>
#include <float.h>
#include <typeinfo>
#include <limits>
#include <algorithm>
#include <vector>

#include <omp.h>

#include <hipsparse.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

//#include <thrust/execution_policy.h>
//#include <thrust/device_vector.h>
//#include <thrust/scan.h>
//#include <thrust/sort.h>

#include "spmspv/csc-spmspv/spmspv.h"
#include "spmspv/class.hpp"
#include "spmspv/mtx.hpp"
#include "spmspv/readsvmdata.hpp"
#include "spmspv/config.h"

#ifndef VALUE_TYPE
#define VALUE_TYPE float
#endif

#ifndef NUM_RUN
#define NUM_RUN 10
#endif
    
#define IS_DOUBLE_ZERO(d)  (abs(d) < DBL_EPSILON)
#define IS_FLOAT_ZERO(d)  (abs(d) < FLT_EPSILON)


void TestCoo2Csr(int m, int mat_nnz,
                 int* coo_row, int* csr_row) {

  int* d_csr_row = NULL;
  int* d_coo_row = NULL;
  
  checkCudaErrors(hipMalloc((void** )&d_csr_row,  (m + 1) * sizeof(int)));
  checkCudaErrors(hipMalloc((void** )&d_coo_row, mat_nnz * sizeof(int)));

  checkCudaErrors(hipMemcpy(d_coo_row, coo_row, mat_nnz * sizeof(int),   
                  hipMemcpyHostToDevice));

  hipsparseHandle_t sparse_handle;
  CUSP_CALL(hipsparseCreate(&sparse_handle));
  hipsparseMatDescr_t descr = 0;
  CUSP_CALL(hipsparseCreateMatDescr(&descr));
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  checkCudaErrors(hipDeviceSynchronize());

  hipEvent_t st, ed;
  float tm = 0;
  hipEventCreate(&st);
  hipEventCreate(&ed);
  hipEventRecord(st, 0);

  CUSP_CALL(hipsparseXcoo2csr(sparse_handle, 
            d_coo_row, mat_nnz, m, d_csr_row, 
            HIPSPARSE_INDEX_BASE_ZERO));

  hipEventRecord(ed, 0);
  hipEventSynchronize(ed);
  //unit: ms.
  hipEventElapsedTime(&tm, st, ed);
  hipEventDestroy(st);
  hipEventDestroy(ed);

  std::cout << "coo2csr time = " << tm <<" ms." << std::endl;

  checkCudaErrors(hipMemcpy(csr_row, d_csr_row, (m + 1) * sizeof(int),   
                 hipMemcpyDeviceToHost));
  
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipFree(d_csr_row));
  checkCudaErrors(hipFree(d_coo_row));
}  

void testHybridSpMV(int m, int n, int mat_nnz, int csr_nnz, int* csr_row, int* csr_col, VALUE_TYPE* csr_val, int* csc_row, int* csc_col, VALUE_TYPE* csc_val, int* x_key, VALUE_TYPE* x_val, VALUE_TYPE* x_dense, VALUE_TYPE* y_dense){
  int err = 0;
  hipError_t err_cuda = hipSuccess;
  
  int device_id = 0;
  hipSetDevice(device_id);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device_id);

  std::cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " 
            << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << std::endl;
  
  int* d_csr_row = NULL;
  int* d_csr_col = NULL;
  VALUE_TYPE* d_csr_val = NULL;
  cudaErrCheck(hipMalloc((void **)&d_csr_row,  (m+1) * sizeof(int)));
  cudaErrCheck(hipMalloc((void **)&d_csr_col,  csr_nnz  * sizeof(int)));
  cudaErrCheck(hipMalloc((void **)&d_csr_val,  csr_nnz  * sizeof(VALUE_TYPE)));
  cudaErrCheck(hipMemcpy(d_csr_row, csr_row, (m+1) * sizeof(int),   
      hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(d_csr_col, csr_col, csr_nnz * sizeof(int),   
      hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(d_csr_val, csr_val, csr_nnz * sizeof(VALUE_TYPE),   
      hipMemcpyHostToDevice));
  
  int csc_nnz = mat_nnz - csr_nnz;
  int* d_csc_row = NULL;
  int* d_csc_col = NULL;
  VALUE_TYPE* d_csc_val = NULL;
  cudaErrCheck(hipMalloc((void **)&d_csc_col,  (n+1) * sizeof(int)));
  cudaErrCheck(hipMalloc((void **)&d_csc_row,  csc_nnz  * sizeof(int)));
  cudaErrCheck(hipMalloc((void **)&d_csc_val,  csc_nnz  * sizeof(VALUE_TYPE)));
  cudaErrCheck(hipMemcpy(d_csc_col, csc_col, (n+1) * sizeof(int), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(d_csc_row, csc_row, csc_nnz * sizeof(int), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(d_csc_val, csc_val, csc_nnz * sizeof(VALUE_TYPE), hipMemcpyHostToDevice));
  
  
  VALUE_TYPE* d_x = NULL;
  cudaErrCheck(hipMalloc((void** )&d_x, n * sizeof(VALUE_TYPE)));
  cudaErrCheck(hipMemcpy(d_x, x_dense, n * sizeof(VALUE_TYPE), hipMemcpyHostToDevice));
  
  int* d_x_key = NULL;
  VALUE_TYPE* d_x_val = NULL;
  int x_nnz = n;//for dense input vector.
  cudaErrCheck(hipMalloc((void** )&d_x_key, (n) * sizeof(int)));
  cudaErrCheck(hipMalloc((void** )&d_x_val, (n) * sizeof(VALUE_TYPE)));
  cudaErrCheck(hipMemcpy(d_x_key, x_key, x_nnz * sizeof(int), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(d_x_val, x_val, x_nnz * sizeof(VALUE_TYPE), hipMemcpyHostToDevice));
  
  int num_ints = (n + sizeof(int) * 8 - 1) / (sizeof(int) * 8);
  int* d_bit_vector = NULL;
  cudaErrCheck(hipMalloc((void** )&d_bit_vector, num_ints * sizeof(int)));

  SpmspvHandle<int, unsigned int, VALUE_TYPE> A(m, n, csr_nnz);
  err = A.InputCSR(csr_nnz, d_csr_row, d_csr_col, d_csr_val);
  
  err = A.set_vector_type(1);//current vector type is densetype.
  err = A.set_x(d_x); //
  err = A.set_bitvector(d_bit_vector, num_ints);//
  
  VALUE_TYPE* d_y = NULL; //result vec.
  cudaErrCheck(hipMalloc((void** )&d_y, m * sizeof(VALUE_TYPE)));
  cudaErrCheck(hipMemset(d_y, 0,  m * sizeof(VALUE_TYPE))); //initialized to zero.
  
  int* d_y_key = NULL; //result vec.
  VALUE_TYPE* d_y_val = NULL; 
  cudaErrCheck(hipMalloc((void** )&d_y_key, m * sizeof(int)));
  cudaErrCheck(hipMalloc((void** )&d_y_val, m * sizeof(VALUE_TYPE)));

  VALUE_TYPE alpha_i = 1.0;

  SpmspvTimer timer;
  
  err = A.holaPreprocess();
  timer.Start();
  for (int i = 0; i < NUM_RUN; i++)
    err = A.holaspmv(alpha_i, d_y);
  err_cuda = hipDeviceSynchronize();
  double holaspmv_time = timer.Stop()/ (double)NUM_RUN;
  std::cout << "holaspmv time = " << holaspmv_time << std::endl; 

  A.holaPostprocess();
  A.Destroy();
  
  
  SpmspvHandle<int, unsigned int, VALUE_TYPE> A2(m, n, csc_nnz);
  err = A2.InputCSC(csc_nnz, d_csc_row, d_csc_col, d_csc_val);
  
  err = A2.set_sparse_x(d_x_key, d_x_val, x_nnz);//
  err = A2.set_x(d_x);//
    
  
  A2.allocPreBuffer();
    
  timer.Start();
  int bin_len = A2.computeBinlenVer2();
  double time = timer.Stop();
  std::cout << "DEBUG: compute bin_len time = " << time << "ms." << std::endl;
    
  int y_nnz = 0;
  cudaErrCheck(hipMemset(d_y, 0,  m * sizeof(VALUE_TYPE))); //initialized to zero.
  timer.Start();
  //err = A.CscBasedNoSortMergeSpmspv_keyval(true, alpha_i, &y_nnz, d_y, d_y_key, d_y_val);
  err = A2.CscBasedNoSortMergeSpmspv(true, alpha_i, &y_nnz, d_y, d_y_key, d_y_val);
  double csc_time = timer.Stop();
  std::cout << "my lb col-spmspv time = " << csc_time << "ms." << std::endl;
  std::cout << "all time = " << csc_time + holaspmv_time << "ms." << std::endl;
   
  hipError_t err_r = hipGetLastError();
    if ( hipSuccess != err_r) {
      printf("cscspmspv() invocate error.\n");
      std::cout << "err code: " << hipGetErrorString(err_r) << std::endl;
      exit(1);
    }
    
  A2.deallocPreBuffer();
  A2.Destroy();
  
  
  if (d_csr_row) cudaErrCheck(hipFree(d_csr_row));
  if (d_csr_col) cudaErrCheck(hipFree(d_csr_col));
  if (d_csr_val) cudaErrCheck(hipFree(d_csr_val));
  
  if (d_x)       hipFree(d_x);
  if (d_x_key)   hipFree(d_x_key);
  if (d_x_val)   hipFree(d_x_val);
  
  if (d_y)       hipFree(d_y);
  if (d_y_key)       hipFree(d_y_key);
  if (d_y_val)       hipFree(d_y_val);

  //return err;
}

int main(int argc, char** argv) {
  int m, n, mat_nnz;
  int* coo_row;
  int* coo_col;
  VALUE_TYPE* coo_val;
  
  int* csr_cluster_row;
  int* csr_col;
  VALUE_TYPE* csr_val;
  
  int* csc_row;
  int* csc_cluster_col;
  VALUE_TYPE* csc_val;

  int* csr_row;
  int* csc_col;

  int ret_csr_nnz = 0;
  // report precision of floating-point
  std::cout << "-------------------------" << std::endl;
  char* precision;
  if (sizeof(VALUE_TYPE) == 4) {
      precision = "32-bit Single Precision (float)";
  } else if (sizeof(VALUE_TYPE) == 8) {
      precision = "64-bit Double Precision (double)";
  } else {
      std::cout << "Wrong precision. Program exit!" << std::endl;
      return 0;
  }
  std::cout << "PRECISION = " << precision << std::endl;
  std::cout << "-------------------------" << std::endl;

  char* file_name;
  if (argc == 2) {
    file_name = argv[1];
  } else {
    std::cout << "Usage: matrix_file_name" << std::endl;
    exit(1);
  }
  std::cout << "-------" << file_name << "---------" << std::endl;

  MTX<VALUE_TYPE> mtx;
  fileToMtxCoo<VALUE_TYPE>(file_name, &mtx, true); 
  m = mtx.rows;
  n = mtx.cols;
  mat_nnz = mtx.nnz;

  printf("m=%d, n=%d, nnz=%d\n", m, n, mat_nnz);
  
  coo_row = mtx.row;
  coo_col = mtx.col;
  coo_val = mtx.data;
  
  COO2Hybrid<VALUE_TYPE>(1, m, n, mat_nnz, coo_row, coo_col, coo_val, &ret_csr_nnz);
  
  std::cout << "ret csr_nnz = " << ret_csr_nnz << std::endl;

  csr_cluster_row = coo_row;
  csr_col = coo_col;
  csr_val = coo_val;

  csc_row = &coo_row[ret_csr_nnz];
  csc_cluster_col = &coo_col[ret_csr_nnz];
  csc_val = &coo_val[ret_csr_nnz];
  
  csr_row = (int* )malloc((m + 1) * sizeof(int));
  CHECK_MALLOC(csr_row);
  
  csc_col = (int* )malloc((n + 1) * sizeof(int));
  CHECK_MALLOC(csc_col);
#if 1  
  TestCoo2Csr(m, ret_csr_nnz, csr_cluster_row, csr_row);
  TestCoo2Csr(n, mat_nnz - ret_csr_nnz, csc_cluster_col, csc_col);
#endif
  int* x_key = (int* )malloc(n * sizeof(int));
  CHECK_MALLOC(x_key);
  VALUE_TYPE* x_val = (VALUE_TYPE* )malloc(n * sizeof(VALUE_TYPE));
  CHECK_MALLOC(x_val);
  VALUE_TYPE* x_dense = (VALUE_TYPE* )malloc(n * sizeof(VALUE_TYPE));
  CHECK_MALLOC(x_dense);
  for(int i=0; i<n; i++) {
    x_dense[i] = (VALUE_TYPE)i;
    x_key[i] = i;
    x_val[i] = (VALUE_TYPE)i;
  }
  VALUE_TYPE* y_dense = (VALUE_TYPE* )malloc(m * sizeof(VALUE_TYPE));
  CHECK_MALLOC(y_dense);
  
  
  testHybridSpMV(m, n, mat_nnz, ret_csr_nnz, csr_row, csr_col, csr_val, csc_row, csc_col, csc_val, 
      x_key, x_val, x_dense, y_dense);
 
  if (mtx.row)   free(mtx.row);
  if (mtx.col)   free(mtx.col);
  if (mtx.data)  free(mtx.data);
  
  if(csr_row) free(csr_row);
  if(csc_col) free(csc_col);
  
  if (x_key)      free(x_key);
  if (x_val)      free(x_val);
  if (x_dense)      free(x_dense);
  if (y_dense)      free(y_dense);
  return 0;
}
